#include "hip/hip_runtime.h"
﻿#include "hip/hip_runtime.h"
#include ""

#include "cryptodata.h"

void UseCuda(int*, const int*, unsigned int, unsigned int);

__global__ void FactorialKernel(int* result, const int* a)
{
	int i = blockIdx.x * blockDim.x + threadIdx.x;

	result[i] = 1;
	for (int j = 1; j <= a[i]; j++)
	{
		result[i] += (j % 2 == 0 ? -1 : 1) * j % 2 + 1;
	}
}

extern "C" __declspec(dllexport) void factorial(int n)
{

}

int main()
{
	read_data_1M("SOLUSDT");

	const int arraySize = 256;
	int a[arraySize];
	int r[arraySize] = { 0 };

	// Input
	for (int i = 0; i < arraySize; i++) {
		a[i] = i + 1;
	}

	// Compute
	UseCuda(r, a, 16, 16);

	// Output
	for (int i = 0; i < arraySize; i++) {
		printf("%d ", r[i]);
	}

	// Free
	hipDeviceReset();
	return 0;
}

/// <summary>
/// block 1개일때 thread max 1024
/// block 256개일때 thread max 256 (=65536)
/// </summary>
/// <param name="c"></param>
/// <param name="a"></param>
/// <param name="size"></param>
/// <returns></returns>
void UseCuda(int* r, const int* a, unsigned int b_size, unsigned int t_size)
{
	int* d_a = 0;
	int* d_r = 0;
	size_t size = b_size * t_size * sizeof(int);

	hipSetDevice(0);
	hipMalloc((void**)&d_r, size);
	hipMalloc((void**)&d_a, size);
	hipMemcpy(d_a, a, size, hipMemcpyHostToDevice);

	FactorialKernel << <b_size, t_size >> > (d_r, d_a);

	hipGetLastError();
	hipDeviceSynchronize();
	hipMemcpy(r, d_r, size, hipMemcpyDeviceToHost);
	hipFree(d_r);
	hipFree(d_a);
}
