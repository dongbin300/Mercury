#include "hip/hip_runtime.h"
﻿#include "hip/hip_runtime.h"
#include ""

#include <stdlib.h>

//int main()
//{
//	//read_data_1M("SOLUSDT");
//
//	const int arraySize = 256;
//	int a[arraySize];
//	int r[arraySize] = { 0 };
//
//	// Input
//	for (int i = 0; i < arraySize; i++) {
//		a[i] = i + 1;
//	}
//
//	// Compute
//	UseCuda(r, a, 16, 16);
//
//	// Output
//	for (int i = 0; i < arraySize; i++) {
//		printf("%d ", r[i]);
//	}
//
//	// Free
//	hipDeviceReset();
//	return 0;
//}

__device__ double average(double* values, int count, int startIndex) {
	double sum = 0;
	for (int i = startIndex; i < startIndex + count; i++) {
		sum += values[i];
	}
	return sum / count;
}

__device__ void ema(double* result, double* values, int length, int period, int startIndex) {
	double alpha = 2.0 / (period + 1);

	for (int i = 0; i < length; i++) {
		if (i < startIndex + period - 1) {
			result[i] = 0;
			continue;
		}

		if (i == startIndex + period - 1) {
			result[i] = average(values, period, startIndex);
			continue;
		}

		result[i] = alpha * values[i] + (1 - alpha) * result[i - 1];
	}
}

__global__ void calculate_ema_kernel(double* r, double* close, int length, int period)
{
	int i = blockIdx.x * blockDim.x + threadIdx.x;

	ema(r, close, length, period, 0);
}

/// <summary>
/// block 1개일때 thread max 1024
/// block 256개일때 thread max 256 (=65536)
/// </summary>
/// <param name="c"></param>
/// <param name="a"></param>
/// <param name="size"></param>
/// <returns></returns>
extern "C" __declspec(dllexport) void use_cuda(double *r, double* close, int length, int period, unsigned int b_size, unsigned int t_size)
{
	double* d_r;
	double* d_close;

	hipSetDevice(0);

	size_t r_size = b_size * t_size * length * sizeof(double);
	size_t close_size = b_size * t_size * length * sizeof(double);
	hipMalloc((void**)&d_r, r_size);
	hipMalloc((void**)&d_close, close_size);
	hipMemcpy(d_close, close, close_size, hipMemcpyHostToDevice);

	calculate_ema_kernel << <b_size, t_size >> > (d_r, d_close, length, period);

	hipGetLastError();
	hipDeviceSynchronize();

	hipMemcpy(r, d_r, r_size, hipMemcpyDeviceToHost);

	hipFree(d_r);
	hipFree(d_close);
}
