#include "hip/hip_runtime.h"
﻿#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>
#include <time.h>

void UseCuda(int*, const int*, unsigned int, unsigned int);

__global__ void addKernel(int* c, const int* a, const int* b)
{
	int i = threadIdx.x;
	c[i] = a[i] * b[i];
}

__global__ void FactorialKernel(int* result, const int* a)
{
	int i = blockIdx.x * blockDim.x + threadIdx.x;

	result[i] = 1;
	for (int j = 1; j <= a[i]; j++)
	{
		result[i] += (j % 2 == 0 ? -1 : 1) * j % 2 + 1;
	}
}

extern "C" __declspec(dllexport) void factorial(int n)
{

}

int main()
{
	const int arraySize = 65536;
	int a[arraySize];
	int r[arraySize] = { 0 };

	// Input
	for (int i = 0; i < arraySize; i++) {
		a[i] = i + 1;
	}

	time_t start_time = clock();
	// Compute
	UseCuda(r, a, 256, 256);
	time_t end_time = clock();

	// Output
	/*for (int i = 0; i < arraySize; i++) {
		printf("%d ", r[i]);
	}*/
	double elapsed_time = ((double)(end_time - start_time)) / CLOCKS_PER_SEC;
	printf("%.3f 초\n", elapsed_time);

	// Free
	hipDeviceReset();
	return 0;
}

/// <summary>
/// block 1개일때 thread max 1024
/// block 256개일때 thread max 256 (=65536)
/// </summary>
/// <param name="c"></param>
/// <param name="a"></param>
/// <param name="size"></param>
/// <returns></returns>
void UseCuda(int* r, const int* a, unsigned int b_size, unsigned int t_size)
{
	int* d_a = 0;
	int* d_r = 0;

	hipSetDevice(0);
	hipMalloc((void**)&d_r, b_size * t_size * sizeof(int));
	hipMalloc((void**)&d_a, b_size * t_size * sizeof(int));
	hipMemcpy(d_a, a, b_size * t_size * sizeof(int), hipMemcpyHostToDevice);

	FactorialKernel << <b_size, t_size >> > (d_r, d_a);

	hipGetLastError();
	hipDeviceSynchronize();
	hipMemcpy(r, d_r, b_size * t_size * sizeof(int), hipMemcpyDeviceToHost);

	hipFree(d_r);
	hipFree(d_a);
}
